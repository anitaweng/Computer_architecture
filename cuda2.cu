#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <iostream>
using namespace std;

/* Mirror operations */
__global__ 
void mirror(uchar4* inputChannel, uchar4* outputChannel, int numRows, int numCols, bool vertical)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int stripe = blockDim.x * gridDim.x;
	for(int i=col; i<numRows*numCols; i=i+stripe)
	{
		 unsigned char Y = 0.299 * inputChannel[i].x + 0.587 * inputChannel[i].y + 0.114 * inputChannel[i].z;
		 if(vertical)
		 	outputChannel[i/numCols*numCols+(numCols-i%numCols)-1] = make_uchar4(Y, Y, Y, 255);
		 else
			outputChannel[(numRows- (i/numCols) -1)*numCols +(i%numCols)] = make_uchar4(Y, Y, Y, 255);
	}
}


uchar4* mirror_ops(uchar4 *d_inputImageRGBA, size_t numRows, size_t numCols, bool vertical)
{
  //Creat Timing Event
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop); 

  //Set reasonable block size (i.e., number of threads per block)
  dim3 blockSize(9);
  //Calculate Grid SIze
  dim3 gridSize(6);
  
  //Calculate number of pixels
  size_t numPixels = numRows * numCols;

  //Allocate Memory Space on Device for output image
  uchar4 *d_outputImageRGBA;
  hipMalloc(&d_outputImageRGBA, sizeof(uchar4) * numPixels);
  
  //start Timer
  hipEventRecord(start, 0);

  //Call mirror kernel.
  mirror<<<gridSize, blockSize>>>(d_inputImageRGBA, d_outputImageRGBA, numRows, numCols, vertical);

  //Stop Timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
  hipDeviceSynchronize(); 
 
  //Initialize memory on host for output uchar4*
  uchar4* h_out;
  h_out = (uchar4*)malloc(sizeof(uchar4) * numPixels);

  //Copy output from device to host
  hipMemcpy(h_out, d_outputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost);
  
  //Cleanup memory on device
  hipFree(d_inputImageRGBA);
  hipFree(d_outputImageRGBA);
  
  //Calculate Elapsed Time
  float elapsedTime; 
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU time = %5.2f ms\n", elapsedTime);

  //return h_out
  return h_out;
}
