#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <boost/program_options.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <vector>
#include <stdio.h>
#include "hip/hip_runtime.h"

using namespace cv;

using namespace std;
using namespace boost::program_options;

size_t numRows, numCols;

//extern uchar4* mirror_ops(uchar4 *d_inputImageRGBA, size_t numRows, size_t numCols, bool vertical);

void loadImageRGBA(string &filename, uchar4 **imagePtr,size_t *numRows, size_t *numCols)
{
    // loading the image
    Mat image = imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
    if(image.empty())
    {
      cerr<<"Failed to load image: "<<filename<<endl;
      exit(1);
    }
    if(image.channels() != 3)
    {
      cerr<<"Image must be color!"<<endl;
      exit(1);
    }
    if(!image.isContinuous())
    {
      cerr<<"Image isn't continuous!"<<endl;
      exit(1);
    }

    // forming a 4-channel(RGBA) image.
    Mat imageRGBA;
    cvtColor(image, imageRGBA, CV_BGR2RGBA);
    //cout<<imageRGBA<<endl<<endl;
    *imagePtr = new uchar4[image.rows * image.cols];
    unsigned char *cvPtr = imageRGBA.ptr<unsigned char>(0);
    for(size_t i = 0; i < image.rows * image.cols; ++i)
    {
      (*imagePtr)[i].x = cvPtr[4*i + 0];
      (*imagePtr)[i].y = cvPtr[4*i + 1];
      (*imagePtr)[i].z = cvPtr[4*i + 2];
      (*imagePtr)[i].w = cvPtr[4*i + 3];
    }
    *numRows = image.rows;
    *numCols = image.cols;
}

void saveImageRGBA(uchar4* image, string &output_filename,size_t numRows, size_t numCols)
{
    // Forming the Mat object from uchar4 array.
    int sizes[2] = {numRows, numCols};
    Mat imageRGBA(2, sizes, CV_8UC4, (void *)image);
    // Converting back to BGR system
    Mat imageOutputBGR;
    cvtColor(imageRGBA, imageOutputBGR, CV_RGBA2BGR);
    //cout<<imageOutputBGR<<endl<<endl;
    // Writing the image
    imwrite(output_filename.c_str(), imageOutputBGR);
}

uchar4* load_image_in_GPU(string filename)
{ 
    // Load the image into main memory
    uchar4 *h_image, *d_in;
    loadImageRGBA(filename, &h_image, &numRows, &numCols);
    // Allocate memory to the GPU
    hipMalloc((void **) &d_in, numRows * numCols * sizeof(uchar4));
    hipMemcpy(d_in, h_image, numRows * numCols * sizeof(uchar4), hipMemcpyHostToDevice);
    // No need to keep this image in RAM now.
    free(h_image);
    return d_in;
}

int main(int argc, char* argv[])
{
    string input_file = "images.jpeg";
    string output_file = "output.jpg";
    string vh = argv[1];
    uchar4 *d_in = load_image_in_GPU(input_file);
    uchar4 *h_out = NULL;
    // Performing the required operation
    bool isVertical = ((vh == "v") ? true:false);
    h_out = mirror_ops(d_in, numRows, numCols, isVertical);
    hipFree(d_in);
    if(h_out != NULL)
      saveImageRGBA(h_out, output_file, numRows, numCols);
    /*uchar4 *g_out = NULL;
    g_out = apply_filter(d_in, numRows, numCols, filter_name); 
    hipFree(d_in);
    if(g_out != NULL)
      saveImageRGBA(g_out, gray, numRows, numCols);*/  
}
